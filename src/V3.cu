#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define INPUT_SIZE    784
#define HIDDEN_SIZE   128
#define OUTPUT_SIZE   10
#define LEARNING_RATE 0.01
#define EPOCHS        5

// Timer
double get_time(clock_t start) {
    return (double)(clock() - start) / CLOCKS_PER_SEC;
}

// Allocate/free host matrix
double** allocateMatrix(int rows, int cols) {
    double** mat = (double**)malloc(rows * sizeof(double*));
    for (int i = 0; i < rows; i++)
        mat[i] = (double*)malloc(cols * sizeof(double));
    return mat;
}
void freeMatrix(double** mat, int rows) {
    for (int i = 0; i < rows; i++)
        free(mat[i]);
    free(mat);
}

// MNIST loaders
double** loadMNISTImages(const char* filename, int numImages) {
    FILE* file = fopen(filename, "rb");
    if (!file) { perror(filename); exit(1); }
    fseek(file, 16, SEEK_SET);
    double** images = allocateMatrix(numImages, INPUT_SIZE);
    for (int i = 0; i < numImages; i++) {
        for (int j = 0; j < INPUT_SIZE; j++) {
            unsigned char pixel;
            if (fread(&pixel, 1, 1, file) != 1) {
                fprintf(stderr, "Error reading image\n");
                exit(1);
            }
            images[i][j] = pixel / 255.0;
        }
    }
    fclose(file);
    return images;
}
double** loadMNISTLabels(const char* filename, int numLabels) {
    FILE* file = fopen(filename, "rb");
    if (!file) { perror(filename); exit(1); }
    fseek(file, 8, SEEK_SET);
    double** labels = allocateMatrix(numLabels, OUTPUT_SIZE);
    for (int i = 0; i < numLabels; i++) {
        unsigned char label;
        if (fread(&label, 1, 1, file) != 1) {
            fprintf(stderr, "Error reading label\n");
            exit(1);
        }
        for (int j = 0; j < OUTPUT_SIZE; j++)
            labels[i][j] = (j == label) ? 1.0 : 0.0;
    }
    fclose(file);
    return labels;
}

// CUDA kernels

__global__ void kernel_forward_hidden(
    const double* __restrict__ d_W1,
    const double* __restrict__ d_b1,
    const double* __restrict__ d_input,
          double*             d_hidden
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < HIDDEN_SIZE) {
        double sum = d_b1[i];
        for (int j = 0; j < INPUT_SIZE; j++)
            sum += d_W1[i * INPUT_SIZE + j] * d_input[j];
        d_hidden[i] = fmax(sum, 0.0);
    }
}


__global__ void kernel_forward_output(
    const double* __restrict__ d_W2,
    const double* __restrict__ d_b2,
    const double* __restrict__ d_hidden,
          double*       d_output
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < OUTPUT_SIZE) {
        double sum = d_b2[i];
        for (int j = 0; j < HIDDEN_SIZE; j++)
            sum += d_W2[i*HIDDEN_SIZE + j] * d_hidden[j];
        d_output[i] = sum;
    }
}

__global__ void softmaxkernel(double* x) {
    __shared__ double exps[OUTPUT_SIZE];
    int tid = threadIdx.x;
    if (tid < OUTPUT_SIZE) {
        exps[tid] = exp(x[tid]);
    }
    __syncthreads();


    double sum = 0.0;
    if (tid == 0) {
        for (int i = 0; i < OUTPUT_SIZE; i++) 
            sum += exps[i];

        for (int i = 0; i < OUTPUT_SIZE; i++) 
            x[i] = exps[i] / sum;
    }
}


__global__ void kernel_compute_output_grad(
    const double* __restrict__ d_output,
    const double* __restrict__ d_target,
          double*             d_grad_output
) {
    __shared__ double shared_output[OUTPUT_SIZE];
    __shared__ double shared_target[OUTPUT_SIZE];

    int i = threadIdx.x;


    if (i < OUTPUT_SIZE) {
        shared_output[i] = d_output[i];
        shared_target[i] = d_target[i];
    }

    __syncthreads();

    if (i < OUTPUT_SIZE)
        d_grad_output[i] = shared_output[i] - shared_target[i];
}


__global__ void kernel_compute_hidden_grad(
    const double* __restrict__ d_W2,
    const double* __restrict__ d_hidden,
    const double* __restrict__ d_grad_output,
          double*             d_grad_hidden
) {
    __shared__ double shared_grad_output[OUTPUT_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;


    if (threadIdx.x < OUTPUT_SIZE) {
        shared_grad_output[threadIdx.x] = d_grad_output[threadIdx.x];
    }
    __syncthreads();

    if (i < HIDDEN_SIZE) {
        double sum = 0.0;
        for (int j = 0; j < OUTPUT_SIZE; j++)
            sum += d_W2[j * HIDDEN_SIZE + i] * shared_grad_output[j];

        d_grad_hidden[i] = (d_hidden[i] > 0.0) * sum;
    }
}


__global__ void kernel_update_W2(
    double* d_W2,
    const double* d_hidden,
    const double* d_grad_output
) {

    int j = blockIdx.x * blockDim.x + threadIdx.x; 
    int i = blockIdx.y * blockDim.y + threadIdx.y; 


    __shared__ double shared_hidden[HIDDEN_SIZE];
    __shared__ double shared_grad_output[OUTPUT_SIZE];

    if (threadIdx.y == 0 && j < HIDDEN_SIZE)
        shared_hidden[j] = d_hidden[j];
    if (threadIdx.x == 0 && i < OUTPUT_SIZE)
        shared_grad_output[i] = d_grad_output[i];

    __syncthreads();  

    if (i < OUTPUT_SIZE && j < HIDDEN_SIZE) {
        int idx = i * HIDDEN_SIZE + j;
        d_W2[idx] -= LEARNING_RATE * shared_grad_output[i] * shared_hidden[j];
    }
}




__global__ void kernel_update_W1(
    double*       d_W1,
    const double* d_input,
    const double* d_grad_hidden
) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;  
    int i = blockIdx.y * blockDim.y + threadIdx.y; 

    __shared__ double shared_input[INPUT_SIZE];
    __shared__ double shared_grad_hidden[HIDDEN_SIZE];


    if (threadIdx.y == 0 && j < INPUT_SIZE)
        shared_input[j] = d_input[j];
    if (threadIdx.x == 0 && i < HIDDEN_SIZE)
        shared_grad_hidden[i] = d_grad_hidden[i];

    __syncthreads();

    if (i < HIDDEN_SIZE && j < INPUT_SIZE) {
        int idx = i * INPUT_SIZE + j;
        d_W1[idx] -= LEARNING_RATE * shared_grad_hidden[i] * shared_input[j];
    }
}






__global__ void kernel_update_b2(
    double*       d_b2,
    const double* d_grad_output
) {
    __shared__ double shared_grad[OUTPUT_SIZE];

    int i = threadIdx.x;

    if (i < OUTPUT_SIZE) {
        shared_grad[i] = d_grad_output[i];
        __syncthreads();
        d_b2[i] -= LEARNING_RATE * shared_grad[i];
    }
}





__global__ void kernel_update_b1(
    double*       d_b1,
    const double* d_grad_hidden
) {
    __shared__ double shared_grad[HIDDEN_SIZE];

    int i = threadIdx.x;

    if (i < HIDDEN_SIZE) {
        shared_grad[i] = d_grad_hidden[i];
        __syncthreads();
        d_b1[i] -= LEARNING_RATE * shared_grad[i];
    }
}





// Network struct
typedef struct {
    double** W1; double** W2;
    double*   b1; double*   b2;
    double* d_W1; double* d_b1;
    double* d_W2; double* d_b2;
    double* d_input; double* d_hidden; double* d_output;
    double* d_target; double* d_grad_output; double* d_grad_hidden;
} NeuralNetwork;

// Create host network
NeuralNetwork* createNetwork() {
    NeuralNetwork* net = (NeuralNetwork*)malloc(sizeof(NeuralNetwork));
    net->W1 = allocateMatrix(HIDDEN_SIZE, INPUT_SIZE);
    net->W2 = allocateMatrix(OUTPUT_SIZE, HIDDEN_SIZE);
    net->b1 = (double*)calloc(HIDDEN_SIZE, sizeof(double));
    net->b2 = (double*)calloc(OUTPUT_SIZE, sizeof(double));
    srand(time(NULL));
    for (int i = 0; i < HIDDEN_SIZE; i++)
        for (int j = 0; j < INPUT_SIZE; j++)
            net->W1[i][j] = ((double)rand()/RAND_MAX)*0.01;
    for (int i = 0; i < OUTPUT_SIZE; i++)
        for (int j = 0; j < HIDDEN_SIZE; j++)
            net->W2[i][j] = ((double)rand()/RAND_MAX)*0.01;
    return net;
}
 
// Allocate device buffers
void allocDeviceNetwork(NeuralNetwork* net) {
    hipMalloc(&net->d_W1, HIDDEN_SIZE * INPUT_SIZE * sizeof(double));
    hipMalloc(&net->d_b1, HIDDEN_SIZE * sizeof(double));
    hipMalloc(&net->d_W2, OUTPUT_SIZE * HIDDEN_SIZE * sizeof(double));
    hipMalloc(&net->d_b2, OUTPUT_SIZE * sizeof(double));

    hipMalloc(&net->d_input, INPUT_SIZE * sizeof(double));
    hipMalloc(&net->d_hidden, HIDDEN_SIZE * sizeof(double));
    hipMalloc(&net->d_output, OUTPUT_SIZE * sizeof(double));

    hipMalloc(&net->d_target, OUTPUT_SIZE * sizeof(double));
    hipMalloc(&net->d_grad_output, OUTPUT_SIZE * sizeof(double));
    hipMalloc(&net->d_grad_hidden, HIDDEN_SIZE * sizeof(double));

}

// Upload host→device
void uploadDeviceNetwork(NeuralNetwork* net) {
    for (int i = 0; i < HIDDEN_SIZE; i++)
        hipMemcpy(net->d_W1 + i*INPUT_SIZE, net->W1[i],
                   INPUT_SIZE*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_b1, net->b1, HIDDEN_SIZE*sizeof(double), hipMemcpyHostToDevice);
    for (int i = 0; i < OUTPUT_SIZE; i++)
        hipMemcpy(net->d_W2 + i*HIDDEN_SIZE, net->W2[i],
                   HIDDEN_SIZE*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(net->d_b2, net->b2, OUTPUT_SIZE*sizeof(double), hipMemcpyHostToDevice);
}

// Training
void train(NeuralNetwork* net, double** images, double** labels, int numImages) {
    clock_t total_start = clock();
    const int blockSize = 128;

    for (int epoch = 0; epoch < EPOCHS; epoch++) {
        clock_t epoch_start = clock();
        double loss = 0.0;
        int correct = 0;

        for (int i = 0; i < numImages; i++) {
            double output[OUTPUT_SIZE];

            // Upload input to device
            hipMemcpy(net->d_input, images[i], INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);

            // FORWARD PASS
            int grid_hidden = (HIDDEN_SIZE + blockSize - 1) / blockSize;
            int grid_output = (OUTPUT_SIZE + blockSize - 1) / blockSize;

            kernel_forward_hidden<<<grid_hidden, blockSize>>>(
                net->d_W1, net->d_b1, net->d_input, net->d_hidden);

            kernel_forward_output<<<grid_output, blockSize>>>(
                net->d_W2, net->d_b2, net->d_hidden, net->d_output);

            softmaxkernel<<<1, OUTPUT_SIZE>>>(net->d_output);

            // BACKWARD PASS
            hipMemcpy(net->d_target, labels[i], OUTPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);

            
            kernel_compute_output_grad<<<grid_output, blockSize>>>(
                net->d_output, net->d_target, net->d_grad_output);

            kernel_compute_hidden_grad<<<grid_hidden, blockSize>>>(
                net->d_W2, net->d_hidden, net->d_grad_output, net->d_grad_hidden);

            dim3 block2d(16, 16);
            dim3 gridW2((HIDDEN_SIZE + 15) / 16, (OUTPUT_SIZE + 15) / 16);
            kernel_update_W2<<<gridW2, block2d>>>(net->d_W2, net->d_hidden, net->d_grad_output);
            

            dim3 gridW1((INPUT_SIZE + 15) / 16, (HIDDEN_SIZE + 15) / 16);
            kernel_update_W1<<<gridW1, block2d>>>(
                net->d_W1, net->d_input, net->d_grad_hidden);

            kernel_update_b2<<<grid_output, blockSize>>>(
                net->d_b2, net->d_grad_output);

            kernel_update_b1<<<grid_hidden, blockSize>>>(
                net->d_b1, net->d_grad_hidden);

            // Copy prediction from device
            hipMemcpy(output, net->d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost);

            // Compute loss and accuracy on host
            for (int k = 0; k < OUTPUT_SIZE; k++)
                loss -= labels[i][k] * log(output[k] + 1e-9);  

            int pred = 0, actual = 0;
            for (int j = 0; j < OUTPUT_SIZE; j++) {
                if (output[j] > output[pred]) pred = j;
                if (labels[i][j] > labels[i][actual]) actual = j;
            }
            if (pred == actual) correct++;
        }

        printf("Epoch %d - Loss: %.4f - Train Accuracy: %.2f%% - Time: %.3fs\n",
            epoch + 1, loss / numImages, (correct / (double)numImages) * 100, get_time(epoch_start));
    }

    printf("Total training time: %.3fs\n", get_time(total_start));
}



// Evaluate
void evaluate(NeuralNetwork* net, double** images, double** labels, int numImages) {
    int correct = 0;
    double output[OUTPUT_SIZE];

    const int blockSize = 128;
    int grid_hidden = (HIDDEN_SIZE + blockSize - 1) / blockSize;
    int grid_output = (OUTPUT_SIZE + blockSize - 1) / blockSize;

    for (int i = 0; i < numImages; i++) {
        hipMemcpy(net->d_input, images[i], INPUT_SIZE * sizeof(double), hipMemcpyHostToDevice);

        kernel_forward_hidden<<<grid_hidden, blockSize>>>(net->d_W1, net->d_b1, net->d_input, net->d_hidden);
        kernel_forward_output<<<grid_output, blockSize>>>(net->d_W2, net->d_b2, net->d_hidden, net->d_output);
        softmaxkernel<<<1, OUTPUT_SIZE>>>(net->d_output);

        hipMemcpy(output, net->d_output, OUTPUT_SIZE * sizeof(double), hipMemcpyDeviceToHost);

        int pred = 0, actual = 0;
        for (int j = 0; j < OUTPUT_SIZE; j++) {
            if (output[j] > output[pred]) pred = j;
            if (labels[i][j] > labels[i][actual]) actual = j;
        }
        if (pred == actual) correct++;
    }
    printf("Test Accuracy: %.2f%%\n", 100.0 * correct / numImages);
}


int main() {
    printf("MNIST Neural Network V3\n");

    // const char* basepath1 = "C:/Users/abgho/Documents/Uni Work/HPC/Project/data/";
    const char* basepath2 = "H:/Github Projects/HPC Project/data/";

    char train_images_path[256], train_labels_path[256];
    char test_images_path[256], test_labels_path[256];

    sprintf(train_images_path, "%strain-images.idx3-ubyte", basepath2);
    sprintf(train_labels_path, "%strain-labels.idx1-ubyte", basepath2);
    sprintf(test_images_path,  "%st10k-images.idx3-ubyte", basepath2);
    sprintf(test_labels_path,  "%st10k-labels.idx1-ubyte", basepath2);

    double** train_images = loadMNISTImages(train_images_path, 60000);
    double** train_labels = loadMNISTLabels(train_labels_path, 60000);
    double** test_images = loadMNISTImages(test_images_path, 10000);
    double** test_labels = loadMNISTLabels(test_labels_path, 10000);

    NeuralNetwork* net = createNetwork();
    allocDeviceNetwork(net);
    uploadDeviceNetwork(net);

    train(net, train_images, train_labels, 60000);
    evaluate(net, test_images, test_labels, 10000);

    freeMatrix(train_images, 60000);
    freeMatrix(train_labels, 60000);
    freeMatrix(test_images,  10000);
    freeMatrix(test_labels,  10000);
    
    return 0;
}
